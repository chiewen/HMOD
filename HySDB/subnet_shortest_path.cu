#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "cuda_guard.cuh"
#include "subnet_shortest_path.cuh"
#include "index_.h"

void SubnetBF::BfOnCpu(int* cells, int cell_num, int start_cell, int start_vertex, int start_edge, int distance, SBfResult* result, int edge_num) {
	int grid_to_result[Index::kCellNum];
	for (int i = 0; i < Index::kCellNum; ++i) {
		grid_to_result[i] = -1;
	}

	int result_idx = 0;
	for (int i = 0; i < cell_num; ++i) {
		grid_to_result[cells[i]] = i;
		for (int j = 0; j < Index::kMaxVerticesPerCell; ++j) {
			auto& r = result[result_idx++];
			r.id_ = Index::grid_[cells[i]].vertex_[j].id_;
			r.distance_ = std::numeric_limits<int>::max();
			r.previous_vertex_id = 0;
		}
	}
	auto& edge = Index::grid_[start_cell].vertex_[start_vertex].edges_[start_edge];
	result[Index::kMaxVerticesPerCell * grid_to_result[edge.to_cell_] + edge.to_vertex_pos_].distance_ = edge.length_ - distance;
	for (int i = 0; i < edge_num; ++i) {
		for (int j = 0; j < cell_num; ++j) {
			auto& c = Index::grid_[cells[j]];
			for (int k = 0; k < Index::kMaxVerticesPerCell; ++k) {
				auto& v = c.vertex_[k];
				int v_dist = result[Index::kMaxVerticesPerCell * grid_to_result[cells[j]] + k].distance_;
				if (v_dist < std::numeric_limits<int>::max()) {
					for (int l = 0; l < Index::kMaxEdgesPerVertex; ++l) {
						auto& e = v.edges_[l];
						if (e.id_ != 0 && grid_to_result[e.to_cell_] != -1) {
							auto& r = result[Index::kMaxVerticesPerCell * grid_to_result[e.to_cell_] + e.to_vertex_pos_];
							if (r.distance_ > v_dist + e.length_) {
								r.distance_ = v_dist + e.length_;
								r.previous_vertex_id = v.id_;
							}
						}
					}
				}
			}
		}
	}
}

__global__ void devBfOnGpu(int* cells, int cell_num, int * grid_to_result, SubnetBF::SBfResult* result, Index::Cell* grid_, int edge_num, int int_max) {
	int cells_per_loop = SubnetBF::kMaxThreadsPerBlock / Index::kMaxVerticesPerCell;
	int loop_num = (cell_num + cells_per_loop - 1) / cells_per_loop;

	for (int i = 0; i < edge_num; ++i) {
		for (int j = 0; j < loop_num; ++j) {
			int current_cell = threadIdx.y + cells_per_loop * j;
			if (current_cell < cell_num) {
				auto& cell = grid_[cells[current_cell]];
				auto& vertex = cell.vertex_[threadIdx.x];
				int v_dist = result[Index::kMaxVerticesPerCell * grid_to_result[cells[current_cell]] + threadIdx.x].distance_;
				if (v_dist < int_max) {
					for (int l = 0; l < Index::kMaxEdgesPerVertex; ++l) {
						auto& e = vertex.edges_[l];
						if (e.id_ != 0 && grid_to_result[e.to_cell_] != -1) {
							auto& r = result[Index::kMaxVerticesPerCell * grid_to_result[e.to_cell_] + e.to_vertex_pos_];
							if (r.distance_ > v_dist + e.length_) {
								r.distance_ = v_dist + e.length_;
								r.previous_vertex_id = vertex.id_;
							}
						}
					}
				}
			}
		}
		__syncthreads();
	}
}

void SubnetBF::BfOnGpu(int* cells, int cell_num, int start_cell, int start_vertex, int start_edge, int distance, SBfResult* result, int edge_num) {
	int grid_to_result[Index::kCellNum];
	for (int i = 0; i < Index::kCellNum; ++i) {
		grid_to_result[i] = -1;
	}

	int result_idx = 0;
	for (int i = 0; i < cell_num; ++i) {
		grid_to_result[cells[i]] = i;
		for (int j = 0; j < Index::kMaxVerticesPerCell; ++j) {
			auto& r = result[result_idx++];
			r.id_ = Index::grid_[cells[i]].vertex_[j].id_;
			r.distance_ = std::numeric_limits<int>::max();
			r.previous_vertex_id = 0;
		}
	}

	auto& edge = Index::grid_[start_cell].vertex_[start_vertex].edges_[start_edge];
	result[Index::kMaxVerticesPerCell * grid_to_result[edge.to_cell_] + edge.to_vertex_pos_].distance_ = edge.length_ - distance;
	
	int * d_cells;
	hipMalloc((void**)&d_cells, sizeof(int) * cell_num);
	hipMemcpy(d_cells, cells, sizeof(int) * cell_num, hipMemcpyHostToDevice);

	int * d_grid_to_result;
	hipMalloc((void**)&d_grid_to_result, sizeof(int) * Index::kCellNum);
	hipMemcpy(d_grid_to_result, grid_to_result, sizeof(int) * Index::kCellNum, hipMemcpyHostToDevice);

	SBfResult * d_result;
	hipMalloc((void**)&d_result, sizeof(SBfResult) * cell_num * Index::kMaxVerticesPerCell);
	hipMemcpy(d_result, result, sizeof(SBfResult) * cell_num * Index::kMaxVerticesPerCell, hipMemcpyHostToDevice);

	dim3 block(Index::kMaxVerticesPerCell, kMaxThreadsPerBlock / Index::kMaxVerticesPerCell);
	devBfOnGpu <<<1, block>>>(d_cells, cell_num, d_grid_to_result, d_result, CudaGuard::pd_grid_, edge_num, std::numeric_limits<int>::max());

	hipMemcpy(result, d_result, sizeof(SBfResult) * cell_num * Index::kMaxVerticesPerCell, hipMemcpyDeviceToHost);

	hipFree(d_grid_to_result);
	hipFree(d_result);
	hipFree(d_cells);
}
