#include "hip/hip_runtime.h"
#include "subnet_shortest_path.cuh"

__global__ void test(A a) {

	a.increment();
	a.print_data();
}

int tt() {

	A h_a;
	h_a.increment();
	h_a.print_data();
	test << <3, 5 >> >(h_a);
	hipDeviceSynchronize();
	h_a.print_data();
	return 0;
}