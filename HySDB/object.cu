#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "object.cuh"
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "index_.h"
#include "cuda_guard.cuh"

Object Objects::objects_[Objects::kTotalObjectNum];

//initialize and step with cpu

//void Objects::Initialize() {
//	for (int i = 0; i < kTotalObjectNum; ++i) {
//		objects_[i].id_ = i;
//		objects_[i].cell_id = rand() % Index::kCellNum;
//
//		objects_[i].vertex_pos_ = rand() % Index::grid_[objects_[i].cell_id].vertex_num;
//		auto& vertex = Index::grid_[objects_[i].cell_id].vertex_[objects_[i].vertex_pos_];
//
//		objects_[i].edge_pos_ = rand() % vertex.edge_num_;
//		auto& edge = vertex.edges_[objects_[i].edge_pos_];
//		objects_[i].edge_id_ = edge.id_;
//
//		objects_[i].position_ = rand() % edge.length_;
//		objects_[i].speed_ = kMinSpeed + rand() % (kMaxSpeed - kMinSpeed);
//	}
//}
//
//void Objects::Step() {
//	for (int i = 0; i < kTotalObjectNum; ++i) {
//		auto& o = objects_[i];
//		o.position_ += o.speed_;
//
//		auto& edge = Index::grid_[o.cell_id].vertex_[o.vertex_pos_].edges_[o.edge_pos_];
//		auto edge_length = edge.length_;
//		while (o.position_ >= edge_length) {
//			o.position_ -= edge_length;
//			o.cell_id = edge.to_cell_;
//			o.vertex_pos_ = edge.to_vertex_pos_;
//			int new_edge_pos = rand() % Index::grid_[o.cell_id].vertex_[o.vertex_pos_].edge_num_;
//			auto& new_edge = Index::grid_[o.cell_id].vertex_[o.vertex_pos_].edges_[new_edge_pos];
//			o.edge_pos_ = new_edge_pos;
//			o.edge_id_ = new_edge.id_;
//		}
//	}
//}

__global__ void devStep(hiprandState* s, Object* objects_, const Index::Cell* __restrict__ grid_) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	auto& o = objects_[idx];
	o.position_ += o.speed_;

	auto& edge = grid_[o.cell_id].vertex_[o.vertex_pos_].edges_[o.edge_pos_];
	auto edge_length = edge.length_;
	while (o.position_ >= edge_length) {
		o.position_ -= edge_length;
		o.cell_id = edge.to_cell_;
		o.vertex_pos_ = edge.to_vertex_pos_;
		int new_edge_pos = hiprand_uniform(s + idx) * grid_[o.cell_id].vertex_[o.vertex_pos_].edge_num_;
		auto& new_edge = grid_[o.cell_id].vertex_[o.vertex_pos_].edges_[new_edge_pos];
		o.edge_pos_ = new_edge_pos;
		o.edge_id_ = new_edge.id_;
	}
}

void Objects::Step() {
	int size = Objects::kTotalObjectNum;

	devStep<<<16, size / 16>>>(CudaGuard::pd_curand_state_, CudaGuard::pd_objects_, CudaGuard::pd_grid_);
	hipDeviceSynchronize();
	hipMemcpy(Objects::objects_, CudaGuard::pd_objects_, size * sizeof(Object), hipMemcpyDeviceToHost);
}


__global__ void devInitialize(hiprandState* s, Object* objects, const Index::Cell* __restrict__ grid_) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(1234, idx, 0, &s[idx]);

	objects[idx].id_ = idx;
	objects[idx].cell_id = hiprand_uniform(s + idx) * Index::kCellNum;

	objects[idx].vertex_pos_ = hiprand_uniform(s + idx) * grid_[objects[idx].cell_id].vertex_num;
	auto& vertex = grid_[objects[idx].cell_id].vertex_[objects[idx].vertex_pos_];

	objects[idx].edge_pos_ = hiprand_uniform(s + idx) * vertex.edge_num_;
	auto& edge = vertex.edges_[objects[idx].edge_pos_];
	objects[idx].edge_id_ = edge.id_;

	objects[idx].position_ = hiprand_uniform(s + idx) * edge.length_;
	objects[idx].speed_ = Objects::kMinSpeed + hiprand_uniform(s + idx) * (Objects::kMaxSpeed - Objects::kMinSpeed);
}

void Objects::Initialize() {
	memset(Objects::objects_, 0, sizeof(Object) * kTotalObjectNum);

	int size = Objects::kTotalObjectNum;

	devInitialize<<<16, size / 16>>>(CudaGuard::pd_curand_state_, CudaGuard::pd_objects_, CudaGuard::pd_grid_);
	hipDeviceSynchronize();
	hipMemcpy(Objects::objects_, CudaGuard::pd_objects_, size * sizeof(Object), hipMemcpyDeviceToHost);
}
