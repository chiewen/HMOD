#include "stdafx.h"
#include "CudaGuard.cuh"
#include <hip/hip_runtime_api.h>
#include ""
#include "index_.h"

Index::Cell* CudaGuard::pd_grid_;
Object* CudaGuard::pd_objects_;
hiprandState* CudaGuard::pd_curand_state_;

CudaGuard::CudaGuard() {
	hipSetDevice(0);

	Index::Initialize();
	hipMalloc((void**)&pd_grid_, sizeof(Index::grid_));
	hipMemcpy(pd_grid_, Index::grid_, sizeof(Index::grid_), hipMemcpyHostToDevice);

	hipMalloc((void**)&pd_objects_, Objects::kTotalObjectNum * sizeof(Object));
	hipMemset(pd_objects_, 0, Objects::kTotalObjectNum * sizeof(Object));

	hipMalloc((void**)&pd_curand_state_, Objects::kTotalObjectNum * sizeof(hiprandState));
}

CudaGuard::~CudaGuard() {
	hipFree(pd_objects_);
	hipFree(pd_grid_);
	hipFree(pd_curand_state_);

	hipDeviceReset();
}
